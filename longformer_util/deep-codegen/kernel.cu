#include "hip/hip_runtime.h"
#include "kernel.h"
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <cstdlib>
#include <stdexcept>

//Matrix A: input features/attention scores/gradients of enwik8/text8, 4 dimension: d1, d2, d3 and d4
#include <stdio.h>
#include "wtime.h"
#include "hip/hip_vector_types.h"

//Matrix B: it can be input features/values/gradients, 4 dimensions
//Matrix C: Output result : attention scores/weighted sum/gradients
//Matrix D: Dilation information: 1 dimension = vector of heads

//https://github.com/allenai/longformer/blob/master/longformer/diagonaled_mm_tvm.py
/* line 156 _diagonaled_mm():  t1 = A, t2 = B, d = D, and r = C
        this should call line 16 = Omid's cuda implementation


 */

// mode 1 : d4c == d4b != d4a && d4a = (Window + WindowUpper + 1) && transposeT1 == 0
// mode 2 : d4c == d4b != d4a && d4a = (Window + WindowUpper + 1) && transposeT1 == 1
// mode 3 : d4c != d4b && d4a = d4b && dfc == (Window + WindowUpper + 1)

const int d1 = 2; //batch size
const int d2 = 4096; //sequence length
const int d3 = 12; //attention head count
const int d4a = 64; //Fourth dimension of matrix A: hidden dimension that means feature length of one token
const int d4b = 64; //Fourth dimension of matrix B: --
const int d4c = 513;
const int aSize = d1*d2*d3*d4a;
const int bSize = d1*d2*d3*d4b;
const int cSize = d1*d2*d3*d4c;

const int d2d3d4c = d2*d3*d4c;
const int d3d4c = d3*d4c;
const int d3d4b = d3*d4b;
const int d4c_half = (d4c+1)/2;
#define  part 16 // power of 2
const int part_1 = part - 1;
const int d4c_part = (d4c + part_1)/part;

const int Window = 256;
const int min_valid = d4c - Window + part_1 ;
const int min_valid_part = min_valid/part;
const int max_invalid = Window - part_1;
const int Dmin = 1;
const int Dmin_minus2 = 2 - Dmin;
const int Dmin2 = 2 * Dmin;
const int Dmin_part = Dmin * part;
const int Dmin_minus2_part = 2 - part * Dmin;
const int Dmin2_part = Dmin2 * part;
const int invalid_idx = Dmin * max_invalid;
const int coef_b = -1 * d3 * min_valid_part * (Dmin2_part + 1); // ai^2 + bi + c = 0,
const int coef_c1 = d3 * (Dmin * Dmin_part + 2 * Dmin -  Dmin * Dmin);
const int d3_d4c_part = d3 * d4c_part;

const int invalid_idx_last = d2 - Dmin * Window;
const int coef_b_last = -1 * d3 * Dmin_part * (2 * d4c_part - 1);
const int coef_a_last = -1 * d3 * Dmin_part;
const int i_last_part = Window / part;

__device__ inline float warp_reduce(float val){
    for(int offset = 16; offset > 0; offset /= 2)
        val+= __shfl_xor_sync (FULL_WARP_MASK,val,offset);
    return val;
}

__device__ inline float warp_reduce(float2 val1){
    float val = val1.x +  val1.y;
    for(int offset = 16; offset > 0; offset /= 2)
        val+= __shfl_xor_sync (FULL_WARP_MASK,val,offset);
    return val;
}

template <int dim_wcount>
__device__ inline float subwarp_reduce(float val){
    for(int offset = 16/dim_wcount; offset > 0; offset /= 2)
        val+= __shfl_down_sync (FULL_WARP_MASK,val,offset, 16);
    return val;
}

void compute_half1_invalid_idx(int* valid_j, int* start_i, int* start_i_last, int* l_size, int* i_last) {
	int i_minusD, i_part, idx_part;
	for (int i = 0; i < invalid_idx + 1; i++) {
		if (i >= Dmin) {
			i_minusD = i - Dmin;
			i_part = i_minusD / Dmin_part;
			idx_part = i_part * Dmin_part;
			valid_j[i] = min_valid_part + i_part + 1;
			start_i[i] = d3 * (min_valid_part * i + ((idx_part * (idx_part + Dmin_minus2_part)) / Dmin2_part) + ((i_minusD % Dmin_part) * i_part) + i_minusD - i_part);
		}
		else {
			valid_j[i] = min_valid_part;
			start_i[i] = d3 * min_valid_part * i;
		}
        //printf("%d: %d %d\n", i, valid_j[i], start_i[i]);
	}
	*i_last = (invalid_idx_last - invalid_idx) * d3 * d4c_part + start_i[invalid_idx];
	*l_size = *i_last + Dmin_part * d3 * ((d4c_part - 1) * i_last_part - (i_last_part * (i_last_part - 1))/2);
	for (int i = 0; i < i_last_part; i++) {
		start_i_last[i] = Dmin_part * d3 * ((d4c_part - 1) * i - (i * (i - 1))/2);
        //printf("%d: %d\n", i, start_i_last[i]);
	}
        
    //printf("%d %d\n", *i_last, *l_size);
}

__global__ void mm4d_gpu_mode1_c_padz(float* a, float* b, float* c, int* dilation, int Window, int Padding, int d2, int d3, int d4a, int d4b, int d4c, int aSize, int bSize, int cSize) {
	int idx_a, idx_b, idx;
	int bx, by, tx, ty, B;

	bx = blockIdx.x;
	by = blockIdx.y;
	tx = threadIdx.x;
	ty = threadIdx.y;
	B = blockDim.x*blockDim.y;
	idx = by*gridDim.x*B + bx*B + ty*blockDim.x + tx;

	int l = idx / (d2 * d3 * d4c);
	int i = (idx / (d3 * d4c)) % d2;
	int q = (idx / d4c) % d3;
	int j = idx % d4c;
	int D = dilation[q];

	if (idx < cSize) {
		c[idx] = 0.0f;
		for (int k = 0; k < d4a; k++) {
			int condition = i + D * (k - Window);
			if (condition >= 0 && condition < d2) {
				idx_a = (((l * d2) + i) * d3 + q) *  d4a + k;
				idx_b = (((l * d2) + i + D * (k - Window)) * d3 + q) *  d4b + j;
				if (idx_a < aSize && idx_b < bSize)	c[idx] += a[idx_a] * b[idx_b];
			}
		}
	}
}

__global__ void mm4d_gpu_mode3_c_padz_old(float* a, float* b, float* c, int* dilation) {
	int idx_a, idx_b, idx;
	int l, i, q, j, D;
	int condition, k, ld2, idx_a_base;
	float sum = 0.0f;

	idx = ((blockIdx.y*gridDim.x + blockIdx.x)*blockDim.x*blockDim.y + threadIdx.y*blockDim.x + threadIdx.x);
	if (idx >= cSize) return;

	i = (idx / d3d4c) % d2;
	q = (idx / d4c) % d3;
	j = idx % d4c;
	D = dilation[q];
	condition = i + D * (j - Window);
	if (condition < 0 || condition >= d2) return;
	l = idx / d2d3d4c;
	ld2 = l * d2;
	idx_a_base = ((ld2 + i) * d3 + q) * d4a;

	for (k = 0; k < d4a; k++) {
		idx_a = idx_a_base + k;
		idx_b = ((ld2 + condition) * d3 + q) * d4b + k;
		sum += a[idx_a] * b[idx_b];
	}
	c[idx] = sum;
}

//Newly written, very slow
__global__ void mm4d_gpu_mode3_pr(float* a, float* b, float* c, int* dilation, int Window, int Padding) {
	int l, i, q, j, D;
	int k, ld2;

    int tid = threadIdx.x;
	int warp_id =  blockIdx.x * 4 + threadIdx.y;
	if (warp_id >= cSize) return;

	i = (warp_id / d3d4c) % d2;//which token sequence
	q = (warp_id / d4c) % d3; //which head
	j = warp_id % d4c;   //which attention result
	D = dilation[q];

	int condition = i + D * (j - Window);
	if (condition < 0 || condition >= d2) return;

	l = warp_id / d2d3d4c;
	ld2 = l * d2;
	
    int idx_a = ((ld2 + i) * d3 + q) * d4a;
    int idx_b = ((ld2 + condition) * d3 + q) * d4b;
    
    float2 a_value = ((float2*)(a + idx_a))[tid];
    float2 b_value = ((float2*)(b + idx_b))[tid];
    float2 sum     = a_value * b_value;
    
    sum.x = warp_reduce(sum);
	if (tid == 0) c[warp_id] = sum.x;
}

__global__ void mm4d_gpu_mode3_pr2(float* a, float* b, float* c, int* dilation, int Window, int Padding) {
	int l, i, q, j, D;
	int k, ld2;

    int tid = threadIdx.x;
	int warp_id =  blockIdx.x * 4 + threadIdx.y;
	if (warp_id >= cSize) return;

	i = (warp_id / d4c) % d2;//which token sequence
	q = blockIdx.y; //(warp_id / d4c) % d3; //which head
	j = warp_id % d4c;   //which attention result
	D = dilation[q];

	int condition = i + D * (j - Window);
	if (condition < 0 || condition >= d2) return;

	l = warp_id / (d2*d4c);
	ld2 = l * d2;
	
    int idx_a = ((ld2 + i) * d3 + q) * d4a;
    int idx_b = ((ld2 + condition) * d3 + q) * d4b;
    
    float2 a_value = ((float2*)(a + idx_a))[tid];
    float2 b_value = ((float2*)(b + idx_b))[tid];
    float2 sum     = a_value * b_value;
    
    sum.x = warp_reduce(sum);
    if (tid == 0) c[((ld2 + i) * d3 + q) * d4c + j] = sum.x;
}

__global__ void mm4d_gpu_mode3_pr3(float* a, float* b, float* c, int* dilation, int Window, int Padding) {
	int l, i, q, j, D;
	int k, ld2;

    /*
    threadIdx.y = [0, 4)     => 4 i
    blockIdx.x = [0, 513)    => j
    blockIdx.y = [0, 4096/4) => 4 rows (i).
    blockIdx.z = [0, 12)     => heads
    */

    int tid = threadIdx.x;
    int abs_i = blockIdx.y* blockDim.y + threadIdx.y; 

    i = abs_i % d2; //which token sequence
    j = blockIdx.x; //which attention result
    q = blockIdx.z; //which head

	D = dilation[q];

	int condition = i + D * (j - Window);
	if (condition < 0 || condition >= d2) return;

	l = abs_i/d2; //which mini-batch
	ld2 = l * d2;
	
    int idx_a = ((ld2 + i) * d3 + q) * d4a;
    int idx_b = ((ld2 + condition) * d3 + q) * d4b;
    
    float2 a_value = ((float2*)(a + idx_a))[tid];
    float2 b_value = ((float2*)(b + idx_b))[tid];
    float2 sum     = a_value * b_value;
    
    sum.x = warp_reduce(sum);
    if (tid == 0) c[((ld2 + i) * d3 + q) * d4c + j] = sum.x;
}

//data-reuse across i dimension
__global__ void mm4d_gpu_mode3_pr4(float* a, float* b, float* c, int* dilation, int Window, int Padding) {
	int l, i, q, j, D;
	int ld2;

    /*
    threadIdx.y = [0, 4)     => 4 i
    blockIdx.x = [0, 513)    => j
    blockIdx.y = [0, 4096/4) => 4 rows (i).
    blockIdx.z = [0, 12)     => heads
    */

    int tid = threadIdx.x;
    int abs_i = blockIdx.y* blockDim.y + threadIdx.y; 

    i = abs_i % d2; //which token sequence
	l = abs_i/d2; //which mini-batch
	ld2 = l * d2;
    j = blockIdx.x*part; //which attention result
    q = blockIdx.z; //which head

	D = dilation[q];

    int j_upper = min(j+part, d4c);
    int idx_a = ((ld2 + i) * d3 + q) * d4a;
    
    int dim_wid = tid / 16;
    int dim_tid = tid % 16;
    float4 a_value = ((float4*)(a + idx_a))[dim_tid];
    
    for (int jj = j + dim_wid; jj < j_upper; jj += 2) {
	    int condition = (i + D * (jj - Window));
	    float4 b_value; 
        if (condition >= 0 && condition < d2) {
            int idx_b = ((ld2 + condition) * d3 + q) * d4b;
            b_value = ((float4*)(b + idx_b))[dim_tid];
        }
        
        float4 sum  = a_value * b_value;
        float  dot  = sum.x + sum.y + sum.z + sum.w;
        dot         = subwarp_reduce<2>(dot);

	    if (condition >= 0 && condition < d2) {
            int index = ((ld2 + i)*d3 + q)*d4c + jj;
            if (dim_tid == 0) c[index] = dot;
        }
    }
}

template <int valid_idx>
inline __device__ void small_exceptions(int idx_a_base, int idx, int thd_id, int d4a, float *a, float *b, float *sum, int idx_b_base) {
	float aa, bb;
	int idx_b;
	unsigned mask;
	for (int k = thd_id; k < d4a ; k += 32) {
		aa = __ldg(&a[idx_a_base + k]);
		for (int p = valid_idx; p < part; p++) {
			idx_b = idx_b_base + p*768 + k;
			bb = __ldg(&b[idx_b]);
			sum[p] += aa * bb;
		}
	}

	for (int offset = 16; offset > 0; offset /= 2) {
		mask = (1 << 2*offset) - 1;
		for (int p = valid_idx; p < part; p++) {
			sum[p] += __shfl_xor_sync(mask, sum[p], offset, 2*offset);
		}
	}
}

template <int valid_idx>
inline __device__ void big_exceptions(int idx_a_base, int idx, int thd_id, int d4a, float *a, float *b, float *sum, int idx_b_base) {
	float aa, bb;
	int idx_b;
	unsigned mask;
	for (int k = thd_id; k < d4a ; k += 32) {
		aa = __ldg(&a[idx_a_base + k]);
		for (int p = 0; p < valid_idx; p++) {
			idx_b = idx_b_base + p*768 + k;
			bb = __ldg(&b[idx_b]);
			sum[p] += aa * bb;
		}
	}

	for (int offset = 16; offset > 0; offset /= 2) {
		mask = (1 << 2*offset) - 1;
		for (int p = 0; p < valid_idx; p++) {
			sum[p] += __shfl_xor_sync(mask, sum[p], offset, 2*offset);
		}
	}
}

__global__ void mm4d_gpu_mode3_c_padz_new(float* a, float* b, float* c, int* dilation, int* valid_j, int* start_i, int* start_i_last, int l_size, int i_last) {
  //int idx_warp = ((blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x);
  //int idx = idx_warp/32;

  int thd_id = threadIdx.x;
  int abs_warp_id = ((blockIdx.y * gridDim.x + blockIdx.x) * blockDim.y + threadIdx.y);
  
  int l, i, q, j_first;
  l = abs_warp_id / l_size;

  int remaining_l = abs_warp_id % l_size;
  int remaining_start = remaining_l - start_i[invalid_idx];
  int remaining_end = remaining_l - i_last;

  if (remaining_start >= 0 && remaining_end < 0) {
    i = remaining_start / d3_d4c_part + invalid_idx;
    q = (remaining_start / d4c_part) % d3;
    j_first = (remaining_start % d4c_part) * part;

  } else if (remaining_end >= 0) {
    int i_part = (coef_b_last + sqrtf(coef_b_last * coef_b_last + 8 * coef_a_last * remaining_end)) / (2 * coef_a_last);
    int valid_j_last = d4c_part - 1 - i_part;
    // remaining_end -= d3 * Dmin_part * ((d4c_part - 1) * i_part - (i_part * (i_part - 1))/2);
    remaining_end -= start_i_last[i_part];
    i = invalid_idx_last + i_part * Dmin_part + remaining_end / (d3 * valid_j_last);
    q = (remaining_end / valid_j_last) % d3;
    j_first = (remaining_end % valid_j_last) * part;

  } else if (remaining_l >= Dmin * d3 * valid_j[0]) {
    i = (coef_b + sqrtf(coef_b * coef_b + 4 * d3 * (coef_c1 + Dmin2_part * remaining_l)))/(2 * d3);

    remaining_start = remaining_l - start_i[i];
    remaining_end = remaining_l - start_i[i+1];

    if (remaining_start < 0) {i = i - 1; remaining_start = remaining_l - start_i[i];}
    else if (remaining_end >= 0) {i = i + 1; remaining_start = remaining_l - start_i[i];}
    remaining_end = remaining_l - start_i[i+1];

    q = remaining_start / valid_j[i];
    j_first = ((remaining_start % valid_j[i]) + d4c_part - valid_j[i]) * part;

  } else {
    i = remaining_l/(d3 * valid_j[0]);
    remaining_start = remaining_l - start_i[i];

    q = (remaining_start / valid_j[0]) % d3;
    j_first = ((remaining_start % valid_j[0]) + d4c_part - valid_j[0]) * part;
  }

  int idx = l * d2d3d4c + i * d3d4c + q * d4c + j_first;
  if (idx >= cSize) return;

  int j_last, D, d4c_label, condition_first, condition_last;
	D = dilation[q];
	condition_first = i + D * (j_first - Window);
	condition_last = condition_first + part_1 * D;
    if (condition_last < 0 || condition_first >= d2) return;
	j_last = j_first + part_1;
	int valid_idx;

	if (j_last < d4c) {
		if (condition_first >= 0 && condition_last < d2) {d4c_label = 3;}
		else if (condition_last < d2) {d4c_label = 1; valid_idx = -1*((condition_first + 1 - D)/D);}
		else {d4c_label = 2, valid_idx = (d2 - condition_first + D - 1)/D;}
	} else {
        d4c_label = 2; valid_idx = d4c - j_first;
    }
	int ld2 = l * d2;
	int idx_a_base = ((ld2 + i) * d3 + q) * d4a;
	int idx_b_base = ((ld2 + condition_first) * d3 + q) * d4b;
	int idx_diff = D * d3d4b;
	int p;
    //if (idx < 513 && idx > 513 + 256) printf("%d %d %d %d %d %d\n", idx_warp, j_first, condition_first, condition_last, idx_diff, Window);

	if (d4c_label == 3) {
		float sum[part] = {0.0f};
		float aa, bb;
		unsigned mask;
		for (int k = thd_id; k < d4a ; k += 32) {
			aa = __ldg(&a[idx_a_base + k]);
			for (p = 0; p < part; p++) {
				bb = __ldg(&b[idx_b_base + p*idx_diff + k]);
				sum[p] += aa * bb;
			}
		}

		for (int offset = 16; offset > 0; offset /= 2) {
			mask = (1 << 2*offset) - 1;
			for (int p = 0; p < part; p++) {
				sum[p] += __shfl_xor_sync(mask, sum[p], offset, 2*offset);
			}
		}
		if (thd_id == 0) {
			for (int p = 0; p < part; p++) {
				c[idx + p] = sum[p];
			}
		}
	}
	else if (d4c_label == 2) {
		float c_idx[part] = {0.0f};
		switch (valid_idx) {
			case 1: big_exceptions<1> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 2: big_exceptions<2> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 3: big_exceptions<3> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 4: big_exceptions<4> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 5: big_exceptions<5> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 6: big_exceptions<6> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 7: big_exceptions<7> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 8: big_exceptions<8> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 9: big_exceptions<9> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 10: big_exceptions<10> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 11: big_exceptions<11> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 12: big_exceptions<12> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 13: big_exceptions<13> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 14: big_exceptions<14> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 15: big_exceptions<15> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 16: big_exceptions<16> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 17: big_exceptions<17> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 18: big_exceptions<18> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 19: big_exceptions<19> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 20: big_exceptions<20> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 21: big_exceptions<21> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 22: big_exceptions<22> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 23: big_exceptions<23> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 24: big_exceptions<24> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 25: big_exceptions<25> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 26: big_exceptions<26> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 27: big_exceptions<27> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 28: big_exceptions<28> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 29: big_exceptions<29> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 30: big_exceptions<30> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 31: big_exceptions<31> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			default: break;
		}
		if (thd_id == 0) {
			for (p = 0; p < valid_idx; p++) {
				c[idx + p] = c_idx[p];
			}
		}
	}
	else if (d4c_label == 1) {
		float c_idx[part] = {0.0f};
		switch (valid_idx) {
			case 1: small_exceptions<1> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 2: small_exceptions<2> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 3: small_exceptions<3> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 4: small_exceptions<4> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 5: small_exceptions<5> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 6: small_exceptions<6> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 7: small_exceptions<7> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 8: small_exceptions<8> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 9: small_exceptions<9> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 10: small_exceptions<10> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 11: small_exceptions<11> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 12: small_exceptions<12> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 13: small_exceptions<13> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 14: small_exceptions<14> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 15: small_exceptions<15> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 16: small_exceptions<16> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 17: small_exceptions<17> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 18: small_exceptions<18> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 19: small_exceptions<19> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 20: small_exceptions<20> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 21: small_exceptions<21> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 22: small_exceptions<22> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 23: small_exceptions<23> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 24: small_exceptions<24> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 25: small_exceptions<25> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 26: small_exceptions<26> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 27: small_exceptions<27> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 28: small_exceptions<28> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 29: small_exceptions<29> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 30: small_exceptions<30> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			case 31: small_exceptions<31> (idx_a_base, idx, thd_id, d4a, a, b, c_idx, idx_b_base); break;
			default: break;
		}
		if (thd_id == 0) {
			for (p = valid_idx; p < part; p++) {
				c[idx + p] = c_idx[p];
			}
		}
	}
}

__global__ void mm4d_gpu_mode3_c_padz(float* a, float* b, float* c, int* dilation, int Window, int Padding, int d2, int d3, int d4a, int d4b, int d4c, int aSize, int bSize, int cSize) {
	int idx_a, idx_b, idx;
	int bx, by, tx, ty, B;
	float aa, bb;

	bx = blockIdx.x;
	by = blockIdx.y;
	tx = threadIdx.x;
	ty = threadIdx.y;
	B = blockDim.x*blockDim.y;
	idx = (by*gridDim.x*B + bx*B + ty*blockDim.x + tx)/32;
	int l = idx / (d2 * d3 * d4c);
	int i = (idx / (d3 * d4c)) % d2;
	int q = (idx / d4c) % d3;
	int j = idx % d4c;
	int D = dilation[q];
	int warp_id = (by*gridDim.x*B + bx*B + ty*blockDim.x + tx)%32;
	float sum = 0.0f;
	if (idx >= cSize) return;
	c[idx] = 0.0f;
	int condition = i + D * (j - Window);
	if (condition < 0 || condition >= d2) return;

	for (int ii = 0; ii < (d4a + abs(32 - d4a) % 32); ii += 32) {
		if (ii + warp_id < aSize) {idx_a = (((l * d2) + i) * d3 + q) *  d4a + ii + warp_id; aa = a[idx_a];} else {aa = 0.0f;}
		if (ii + warp_id < bSize) {idx_b = (((l * d2) + i + D * (j - Window)) * d3 + q) *  d4b + ii + warp_id; bb = b[idx_b];} else {bb = 0.0f;}
		sum += aa * bb;
	}
	__syncwarp();
	for (int offset = 16; offset > 0; offset /= 2) {
		sum += __shfl_xor_sync(0xffffffff, sum, offset, 32);
	}
	if (warp_id == 0) {
		c[idx] = sum;
	}
}

__global__ void mm4d_gpu_mode1_c(float* a, float* b, float* c, int* dilation, int Window, int Padding, int d2, int d3, int d4a, int d4b, int d4c, int aSize, int bSize, int cSize) {
	int idx_a, idx_b, idx;
	int bx, by, tx, ty, B;

	bx = blockIdx.x;
	by = blockIdx.y;
	tx = threadIdx.x;
	ty = threadIdx.y;
	B = blockDim.x*blockDim.y;
	idx = by*gridDim.x*B + bx*B + ty*blockDim.x + tx;

	int l = idx / (d2 * d3 * d4c);
	int i = (idx / (d3 * d4c)) % d2;
	int q = (idx / d4c) % d3;
	int j = idx % d4c;
	int D = dilation[q];

	if (idx < cSize) {
		c[idx] = 0.0f;
		for (int k = 0; k < d4a; k++) {
			int condition = i + D * (k - Window);
			if (condition >= 0 && condition < d2) {
				idx_a = (((l * d2) + i) * d3 + q) *  d4a + k;
				idx_b = (((l * d2) + i + D * (k - Window)) * d3 + q) *  d4b + j;
				if (idx_a < aSize && idx_b < bSize)	c[idx] += a[idx_a] * b[idx_b];
			}
			else {
				c[idx] += Padding;
			}
		}
	}
}

__global__ void mm4d_gpu_mode2_c(float* a, float* b, float* c, int* dilation, int Window, int WindowUpper, int d2, int d3, int d4a, int d4b, int d4c, int aSize, int bSize, int cSize) {
	int idx_a, idx_b, idx;
	int bx, by, tx, ty, B;

	bx = blockIdx.x;
	by = blockIdx.y;
	tx = threadIdx.x;
	ty = threadIdx.y;
	B = blockDim.x*blockDim.y;
	idx = by*gridDim.x*B + bx*B + ty*blockDim.x + tx;

	int l = idx / (d2 * d3 * d4c);
	int i = (idx / (d3 * d4c)) % d2;
	int q = (idx / d4c) % d3;
	int j = idx % d4c;
	int D = dilation[q];

	if (idx < cSize) {
		c[idx] = 0.0f;
		for (int k = 0; k < d4a; k++) {
			int condition = i + D * (k - WindowUpper);
			if (condition >= 0 && condition < d2) {
				idx_a = (((l * d2) + i + D * (k - WindowUpper)) * d3 + q) *  d4a + WindowUpper + Window - k;
				idx_b = (((l * d2) + i + D * (k - WindowUpper)) * d3 + q) *  d4b + j;
				if (idx_a < aSize && idx_b < bSize)	c[idx] += a[idx_a] * b[idx_b];
			}
		}
	}
}

__global__ void mm4d_gpu_mode3_c(float* a, float* b, float* c, int* dilation, int Window, int Padding, int d2, int d3, int d4a, int d4b, int d4c, int aSize, int bSize, int cSize) {
	int idx_a, idx_b, idx;
	int bx, by, tx, ty, B;

	bx = blockIdx.x;
	by = blockIdx.y;
	tx = threadIdx.x;
	ty = threadIdx.y;
	B = blockDim.x*blockDim.y;
	idx = by*gridDim.x*B + bx*B + ty*blockDim.x + tx;

	int l = idx / (d2 * d3 * d4c);
	int i = (idx / (d3 * d4c)) % d2;
	int q = (idx / d4c) % d3;
	int j = idx % d4c;
	int D = dilation[q];

	if (idx < cSize) {
		c[0] = 0.0f;
		int condition = i + D * (j - Window);
		for (int k = 0; k < d4a; k++) {
			if (condition >= 0 && condition < d2) {
				idx_a = (((l * d2) + i) * d3 + q) *  d4a + k;
				idx_b = (((l * d2) + i + D * (j - Window)) * d3 + q) *  d4b + k;
				if (idx_a < aSize && idx_b < bSize)	c[idx] += a[idx_a] * b[idx_b];
			}
			else {
				c[idx] += Padding;
			}
		}
	}
}

__global__ void mm4d_gpu_mode1_padz(float* a, float* b, float* c, int* dilation, int Window, int Padding, int d2, int d3, int d4a, int d4b, int d4c, int aSize, int bSize, int cSize) {
	int idx_a, idx_b, idx;
	int bx, by, tx, ty, B;

	bx = blockIdx.x;
	by = blockIdx.y;
	tx = threadIdx.x;
	ty = threadIdx.y;
	B = blockDim.x*blockDim.y;
	idx = bx*gridDim.y*B + by*B + tx*blockDim.y + ty;

	int l = idx / (d2 * d3 * d4c);
	int i = (idx / (d3 * d4c)) % d2;
	int q = (idx / d4c) % d3;
	int j = idx % d4c;
	int D = dilation[q];

	if (idx < cSize) {
		c[idx] = 0.0f;
		for (int k = 0; k < d4a; k++) {
			int condition = i + D * (k - Window);
			if (condition >= 0 && condition < d2) {
				idx_a = (((l * d2) + i) * d3 + q) *  d4a + k;
				idx_b = (((l * d2) + i + D * (k - Window)) * d3 + q) *  d4b + j;
				if (idx_a < aSize && idx_b < bSize)	c[idx] += a[idx_a] * b[idx_b];
			}
		}
	}
}

__global__ void mm4d_gpu_mode3_padz(float* a, float* b, float* c, int* dilation, int Window, int Padding, int d2, int d3, int d4a, int d4b, int d4c, int aSize, int bSize, int cSize) {
	int idx_a, idx_b, idx;
	int bx, by, tx, ty, B;

	bx = blockIdx.x;
	by = blockIdx.y;
	tx = threadIdx.x;
	ty = threadIdx.y;
	B = blockDim.x*blockDim.y;
	idx = bx*gridDim.y*B + by*B + tx*blockDim.y + ty;

	int l = idx / (d2 * d3 * d4c);
	int i = (idx / (d3 * d4c)) % d2;
	int q = (idx / d4c) % d3;
	int j = idx % d4c;
	int D = dilation[q];

	if (idx < cSize) {
		c[0] = 0.0f;
		int condition = i + D * (j - Window);
		for (int k = 0; k < d4a; k++) {
			if (condition >= 0 && condition < d2) {
				idx_a = (((l * d2) + i) * d3 + q) * d4a + k;
				idx_b = (((l * d2) + i + D * (j - Window)) * d3 + q) *  d4b + k;
				if (idx_a < aSize && idx_b < bSize)	c[idx] += a[idx_a] * b[idx_b];
			}
		}
	}
}

__global__ void mm4d_gpu_mode1(float* a, float* b, float* c, int* dilation, int Window, int Padding, int d2, int d3, int d4a, int d4b, int d4c, int aSize, int bSize, int cSize) {
	int idx_a, idx_b, idx;
	int bx, by, tx, ty, B;

	bx = blockIdx.x;
	by = blockIdx.y;
	tx = threadIdx.x;
	ty = threadIdx.y;
	B = blockDim.x*blockDim.y;
	idx = bx*gridDim.y*B + by*B + tx*blockDim.y + ty;

	int l = idx / (d2 * d3 * d4c);
	int i = (idx / (d3 * d4c)) % d2;
	int q = (idx / d4c) % d3;
	int j = idx % d4c;
	int D = dilation[q];

	if (idx < cSize) {
		c[idx] = 0.0f;
		for (int k = 0; k < d4a; k++) {
			int condition = i + D * (k - Window);
			if (condition >= 0 && condition < d2) {
				idx_a = (((l * d2) + i) * d3 + q) *  d4a + k;
				idx_b = (((l * d2) + i + D * (k - Window)) * d3 + q) *  d4b + j;
				if (idx_a < aSize && idx_b < bSize)	c[idx] += a[idx_a] * b[idx_b];
			}
			else {
				c[idx] += Padding;
			}
		}
	}
}

__global__ void mm4d_gpu_mode2(float* a, float* b, float* c, int* dilation, int Window, int WindowUpper, int d2, int d3, int d4a, int d4b, int d4c, int aSize, int bSize, int cSize) {
	int idx_a, idx_b, idx;
	int bx, by, tx, ty, B;

	bx = blockIdx.x;
	by = blockIdx.y;
	tx = threadIdx.x;
	ty = threadIdx.y;
	B = blockDim.x*blockDim.y;
	idx = bx*gridDim.y*B + by*B + tx*blockDim.y + ty;

	int l = idx / (d2 * d3 * d4c);
	int i = (idx / (d3 * d4c)) % d2;
	int q = (idx / d4c) % d3;
	int j = idx % d4c;
	int D = dilation[q];

	if (idx < cSize) {
		c[idx] = 0.0f;
		for (int k = 0; k < d4a; k++) {
			int condition = i + D * (k - WindowUpper);
			if (condition >= 0 && condition < d2) {
				idx_a = (((l * d2) + i + D * (k - WindowUpper)) * d3 + q) *  d4a + WindowUpper + Window - k;
				idx_b = (((l * d2) + i + D * (k - WindowUpper)) * d3 + q) *  d4b + j;
				if (idx_a < aSize && idx_b < bSize)	c[idx] += a[idx_a] * b[idx_b];
			}
		}
	}
}

__global__ void mm4d_gpu_mode3(float* a, float* b, float* c, int* dilation, int Window, int Padding, int d2, int d3, int d4a, int d4b, int d4c, int aSize, int bSize, int cSize) {
	int idx_a, idx_b, idx;
	int bx, by, tx, ty, B;

	bx = blockIdx.x;
	by = blockIdx.y;
	tx = threadIdx.x;
	ty = threadIdx.y;
	B = blockDim.x*blockDim.y;
	idx = bx*gridDim.y*B + by*B + tx*blockDim.y + ty;

	int l = idx / (d2 * d3 * d4c);
	int i = (idx / (d3 * d4c)) % d2;
	int q = (idx / d4c) % d3;
	int j = idx % d4c;
	int D = dilation[q];

	if (idx < cSize) {
		c[0] = 0.0f;
		int condition = i + D * (j - Window);
		for (int k = 0; k < d4a; k++) {
			if (condition >= 0 && condition < d2) {
				idx_a = (((l * d2) + i) * d3 + q) * d4a + k;
				idx_b = (((l * d2) + i + D * (j - Window)) * d3 + q) *  d4b + k;
				if (idx_a < aSize && idx_b < bSize)	c[idx] += a[idx_a] * b[idx_b];
			}
			else {
				c[idx] += Padding;
			}
		}
	}
}

void mm4d_cpu_mode1(float* a, float* b, float* c, int* dilation, int Window, int Padding, int d2, int d3, int d4a, int d4b, int d4c, int aSize, int bSize, int cSize) {
	int idx_a, idx_b, idx;

	for (idx = 0; idx < cSize; idx++) {
		int l = idx / (d2 * d3 * d4c);
		int i = (idx / (d3 * d4c)) % d2;
		int q = (idx / d4c) % d3;
		int j = idx % d4c;
		int D = dilation[q];
		c[idx] = 0.0f;

		for (int k = 0; k < d4a; k++) {
			int condition = i + D * (k - Window);
			if (condition >= 0 && condition < d2) {
				idx_a = (((l * d2) + i) * d3 + q) *  d4a + k;
				idx_b = (((l * d2) + i + D * (k - Window)) * d3 + q) *  d4b + j;
				if (idx_a < aSize && idx_b < bSize)	c[idx] += a[idx_a] * b[idx_b];
			}
			else {
				c[idx] += Padding;
			}
		}
	}
}

void mm4d_cpu_mode2(float* a, float* b, float* c, int* dilation, int Window, int WindowUpper, int d2, int d3, int d4a, int d4b, int d4c, int aSize, int bSize, int cSize) {
	int idx_a, idx_b, idx;

	for (idx = 0; idx < cSize; idx++) {
		int l = idx / (d2 * d3 * d4c);
		int i = (idx / (d3 * d4c)) % d2;
		int q = (idx / d4c) % d3;
		int j = idx % d4c;
		int D = dilation[q];
		c[idx] = 0.0f;

		for (int k = 0; k < d4a; k++) {
			int condition = i + D * (k - WindowUpper);
			if (condition >= 0 && condition < d2) {
				idx_a = (((l * d2) + i + D * (k - WindowUpper)) * d3 + q) *  d4a + WindowUpper + Window - k;
				idx_b = (((l * d2) + i + D * (k - WindowUpper)) * d3 + q) *  d4b + j;
				if (idx_a < aSize && idx_b < bSize)	c[idx] += a[idx_a] * b[idx_b];
			}
		}
	}
}

void mm4d_cpu_mode3(float* a, float* b, float* c, int* dilation, int Window, int Padding, int d2, int d3, int d4a, int d4b, int d4c, int aSize, int bSize, int cSize) {
	int idx_a, idx_b, idx;

	for (idx = 0; idx < cSize; idx++) {
		int l = idx / (d2 * d3 * d4c);
		int i = (idx / (d3 * d4c)) % d2;
		int q = (idx / d4c) % d3;
		int j = idx % d4c;
		int D = dilation[q];
		c[idx] = 0.0f;

		int condition = i + D * (j - Window);
		for (int k = 0; k < d4a; k++) {
			if (condition >= 0 && condition < d2) {
				idx_a = (((l * d2) + i) * d3 + q) * d4a + k;
				idx_b = (((l * d2) + i + D * (j - Window)) * d3 + q) *  d4b + k;
				if (idx_a < aSize && idx_b < bSize)	c[idx] += a[idx_a] * b[idx_b];
			}
			else {
				c[idx] += Padding;
			}
		}
	}
}

void lformerMM_original(array4d_t<float>& input1, array4d_t<float>& input2, array4d_t<float>& output1, array1d_t<int>& dilation, array1d_t<int>& params, bool GPU){
	int* d = dilation.data_ptr;
	float* a = input1.data_ptr, *b = input2.data_ptr, *c = output1.data_ptr;
	int d1 = output1.last_count, d2 = output1.matrix_count, d3 = output1.row_count;
	int d4a = input1.col_count, d4b = input2.col_count, d4c = output1.col_count;
	int aSize = d1*d2*d3*d4a, bSize = d1*d2*d3*d4b, cSize = d1*d2*d3*d4c;
	int Window = params.data_ptr[0], WindowUpper = params.data_ptr[1], Padding = params.data_ptr[2], transposeT1 = params.data_ptr[3], coalesced = params.data_ptr[4];
	printf("params: %d %d %d %d %d\n", Window, WindowUpper, Padding, transposeT1, coalesced);

	dim3 blockSize(16, 16);
	dim3 gridSize((d1 * d2 + blockSize.x - 1) / blockSize.x, (d3 * d4c + blockSize.y - 1) / blockSize.y);

	//mode 1 and mode 3 are for forward
	//mode 2 and mode 3 are for backward
	if (d4c == d4b) {//mode 1 or mode 2
		if (transposeT1 == 0) {//mode 1: can be called in forward and backward
			if (!GPU) mm4d_cpu_mode1(a, b, c, d, Window, Padding, d2, d3, d4a, d4b, d4c, aSize, bSize, cSize);
			else if (coalesced == 1 && Padding == 0) mm4d_gpu_mode1_c_padz <<<gridSize, blockSize >>>(a, b, c, d, Window, Padding, d2, d3, d4a, d4b, d4c, aSize, bSize, cSize);
			else if (coalesced == 1) mm4d_gpu_mode1_c <<<gridSize, blockSize >>>(a, b, c, d, Window, Padding, d2, d3, d4a, d4b, d4c, aSize, bSize, cSize);
			else if (Padding == 0) mm4d_gpu_mode1_padz <<<gridSize, blockSize >>>(a, b, c, d, Window, Padding, d2, d3, d4a, d4b, d4c, aSize, bSize, cSize);
			else mm4d_gpu_mode1 <<<gridSize, blockSize >>>(a, b, c, d, Window, Padding, d2, d3, d4a, d4b, d4c, aSize, bSize, cSize);
		}
		else {// mode 2: called during gradient back-propagation
			if (!GPU) mm4d_cpu_mode2(a, b, c, d, Window, WindowUpper, d2, d3, d4a, d4b, d4c, aSize, bSize, cSize);
			else if (coalesced == 1) mm4d_gpu_mode2_c <<<gridSize, blockSize >>>(a, b, c, d, Window, WindowUpper, d2, d3, d4a, d4b, d4c, aSize, bSize, cSize);
			else mm4d_gpu_mode2 <<<gridSize, blockSize >>>(a, b, c, d, Window, WindowUpper, d2, d3, d4a, d4b, d4c, aSize, bSize, cSize);
		}
	}
	else {//mode 3: can be called in forward and backward
		if (!GPU) mm4d_cpu_mode3(a, b, c, d, Window, Padding, d2, d3, d4a, d4b, d4c, aSize, bSize, cSize);
		else if (coalesced == 1 && Padding == 0) mm4d_gpu_mode3_c_padz <<<gridSize, blockSize>>>(a, b, c, d, Window, Padding, d2, d3, d4a, d4b, d4c, aSize, bSize, cSize);
		else if (coalesced == 1) mm4d_gpu_mode3_c <<<gridSize, blockSize>>>(a, b, c, d, Window, Padding, d2, d3, d4a, d4b, d4c, aSize, bSize, cSize);
		else if (Padding == 0) mm4d_gpu_mode3_padz <<<gridSize, blockSize>>>(a, b, c, d, Window, Padding, d2, d3, d4a, d4b, d4c, aSize, bSize, cSize);
		else mm4d_gpu_mode3 <<<gridSize, blockSize>>>(a, b, c, d, Window, Padding, d2, d3, d4a, d4b, d4c, aSize, bSize, cSize);
	}
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) printf("CUDA error: %s\n", hipGetErrorString(cudaStatus));
}

void lformerMM(array4d_t<float>& input1, array4d_t<float>& input2, array4d_t<float>& output1, array1d_t<int>& dilation, array1d_t<int>& params, bool GPU){
	int* d = dilation.data_ptr;
	float* a = input1.data_ptr; 
    float* b = input2.data_ptr;
    float* c = output1.data_ptr;
	int Window = params.data_ptr[0], WindowUpper = params.data_ptr[1], Padding = params.data_ptr[2], transposeT1 = params.data_ptr[3], coalesced = params.data_ptr[4];
	
    //printf("params: %d %d %d %d %d\n", Window, WindowUpper, Padding, transposeT1, coalesced);

    /*
	int *valid_j = (int *)malloc((invalid_idx +1) * sizeof(int));  // valid_j : valid numbers of j
	int *start_i = (int *)malloc((invalid_idx +1) * sizeof(int));  // start_i : index of first element
	int *start_i_last = (int *)malloc(i_last_part * sizeof(int));

	int *v_j, *s_i, *s_i_l;
    int l_size, i_last; // l_size: valid elements in i, q, j
	hipMalloc(&v_j, (invalid_idx +1) * sizeof(int));
	hipMalloc(&s_i, (invalid_idx +1) * sizeof(int));
	hipMalloc(&s_i_l, i_last_part * sizeof(int));
	compute_half1_invalid_idx(valid_j, start_i, start_i_last, &l_size, &i_last);
	hipMemcpy(v_j, valid_j, (invalid_idx +1) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(s_i, start_i, (invalid_idx +1) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(s_i_l, start_i_last, i_last_part * sizeof(int), hipMemcpyHostToDevice);
    */

	dim3 blockSize(8, 8);
	dim3 gridSize((d1 * d2 + blockSize.x - 1) / blockSize.x, (d3 * d4c + blockSize.y - 1) / blockSize.y);
	
    dim3 blocks(32, 4);
	dim3 gridSize_c(((32/part)*(d1 * d2 + blocks.x - 1) / blocks.x), (d3 * d4c + blocks.y - 1) / blocks.y);
	//dim3 grids ((d1*d2*d4c + 3) /4, d3);
	//dim3 grids (d4c, d1*((d2 + 3)/4), d3);
	dim3 grids ((d4c + part_1)/part, d1*((d2 + 3)/4), d3);
	//printf("%d, %d %d, %d, %d %d %d %d %d\n", d1, d2, d3, d4a, d4b, d4c, grids.x, gridSize_c.x, gridSize_c.y);
	//double start = mywtime();

	if (d4c != d4b) { //mode3
		if (coalesced == 1)
            //mm4d_gpu_mode3_c_padz_new<<<gridSize_c, blocks>>>(a, b, c, d, v_j, s_i, s_i_l, l_size, i_last);
            mm4d_gpu_mode3_pr4<<<grids, blocks>>>(a, b, c, d, Window, Padding);
		else
            mm4d_gpu_mode3_c_padz_old<<<gridSize, blockSize>>>(a, b, c, d);
	}
	else {
		throw std::invalid_argument("coalesced kernel for mode 1 and 2 is not implemented.");
	}

	/*
 	hipDeviceSynchronize();
	double end = mywtime();
	printf("cuda time = %f\n", end - start);
	hipFree(s_i); hipFree(s_i_l); hipFree(v_j);
	delete[] start_i; delete[] start_i_last, delete[] valid_j;
    */
}
